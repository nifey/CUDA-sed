#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <strings.h>
#include "nfa.h"

int* copyNfasetToGPU(NFAset* nfaset){
    int* dnfaset;
    int nfadatalen = nfaset->nfadata[(nfaset->n_nfa-1)*5+0] + nfaset->nfadata[(nfaset->n_nfa-1)*5+1];
    hipMalloc(&dnfaset, sizeof(int)*nfadatalen);
    hipMemcpy(dnfaset, nfaset->nfadata, sizeof(int)*nfadatalen, hipMemcpyHostToDevice);
    return dnfaset;
}

char* copyReplacementStringsToGPU(char* strings){
    char* dstrings;
    hipMalloc(&dstrings, sizeof(char)*strlen(strings));
    hipMemcpy(dstrings, strings, sizeof(char)*strlen(strings), hipMemcpyHostToDevice);
    return dstrings;
}

void copyLinesToGPU(int* linestart,int* linelen, char* input_buffer, int* blockdatalen, int** dlinestart, int** dlinelen, char** dbuffer){
  hipMalloc(dlinestart, sizeof(int)*NUM_BLOCKS*THREADS_PER_BLOCK);
  hipMemcpy(*dlinestart, linestart, sizeof(int)*NUM_BLOCKS*THREADS_PER_BLOCK, hipMemcpyHostToDevice);
  hipMalloc(dlinelen, sizeof(int)*NUM_BLOCKS*THREADS_PER_BLOCK);
  hipMemcpy(*dlinelen, linelen, sizeof(int)*NUM_BLOCKS*THREADS_PER_BLOCK, hipMemcpyHostToDevice);

  hipMalloc(dbuffer, sizeof(char)*NUM_BLOCKS*THREADS_PER_BLOCK*MAX_LINE_LENGTH);
  char *current = input_buffer;
  for(int i=0; i<NUM_BLOCKS; i++){
    hipMemcpy(*dbuffer + (i* MAX_LINE_LENGTH * THREADS_PER_BLOCK), current, sizeof(char) * blockdatalen[i], hipMemcpyHostToDevice);
    current += blockdatalen[i];
  }
}

void copyLinesBackAndPrint(char* dbuffer, int num_lines, int lastnewline){
  char* buffer = (char* ) malloc (sizeof(char)*NUM_BLOCKS*THREADS_PER_BLOCK*MAX_LINE_LENGTH);
  hipMemcpy(buffer, dbuffer, sizeof(char)*NUM_BLOCKS*THREADS_PER_BLOCK*MAX_LINE_LENGTH, hipMemcpyDeviceToHost);
	int i;
  for(i=0; i<num_lines-1; i++){
    if(strlen(buffer+i*MAX_LINE_LENGTH) != 0){
      printf("%s\n", buffer+i*MAX_LINE_LENGTH);
    } else {
			printf("\n");
		}
  }
	//The last line might not have a newline character
  if(strlen(buffer+i*MAX_LINE_LENGTH) != 0){
		if(lastnewline == 1){
				printf("%s\n", buffer+i*MAX_LINE_LENGTH);
		} else {
				printf("%s", buffer+i*MAX_LINE_LENGTH);
		}
	} else {
		printf("\n");
  }
  hipFree(dbuffer);
  free(buffer);
}

__host__ __device__ void add_to_next_states(int* nfadata, int state, int* next_states, int* n_next_states){
	int split_index, split_len = 0;
	int n_char_entry = (nfadata[3] - 2)/3;
	for(int i=0; i<n_char_entry; i++){
		if(nfadata[2+ i*3 + 0] == SPLIT_STATE){
			split_index = nfadata[2+ i*3 + 1];
			split_len = nfadata[2+ i*3 + 2];
		}
	}
	int split_states[2], flag = 0;
	for(int i=0; i<split_len; i++){
		if(nfadata[split_index + i*2 + 0] == state){
			split_states[flag++] = nfadata[split_index + i*2 + 1];
		}
		if(flag==2)
			break;
	}
	if(flag == 0){
		split_states[flag++] = state;
	}
	for(int k = 0; k < flag; k++){
		for(int i = 0; i< *n_next_states; i++){
			if(next_states[i] == split_states[k])
				return;
		}
		next_states[*n_next_states] = split_states[k];
		*n_next_states = *n_next_states + 1;
	}
}

__host__ __device__ void make_nfa_transition(int* nfadata, char character, int state, int* next_states, int* n_next_states){
	int n_char_entry = (nfadata[3] - 2)/3;
	int any_index, any_len = 0;
	for(int i=0; i<n_char_entry; i++){
		if (nfadata[2+ i*3 + 0] == MATCH_ANY){
			any_index = nfadata[2 + i*3 + 1];
			any_len = nfadata[2 + i*3 + 2];
		}
	}
	for(int i=0; i<any_len; i++){
		if(nfadata[any_index + i*2 + 0] == state){
			add_to_next_states(nfadata, nfadata[any_index + i*2 + 1], next_states, n_next_states);
		}
	}
	for(int i=0; i<n_char_entry; i++){
		if(nfadata[2+ i*3 + 0] == (int) character){
			for(int j=0; j<nfadata[2+i*3 + 2]; j++){
				if(nfadata[nfadata[2 + i*3 + 1] + j*2 + 0] == state){
					add_to_next_states(nfadata, nfadata[nfadata[2 + i*3 + 1] + j*2 + 1], next_states, n_next_states);
				}
			}
			break;
		}
	}
}

__global__ void processLines(int* nfaset,int n_nfa, char* replacement_strings,int* dlinestart,int* dlinelen,char* dbuffer){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    char *line, *new_line;
    line = (char*) malloc (sizeof(char)*MAX_LINE_LENGTH);
    new_line = (char*) malloc (sizeof(char)*MAX_LINE_LENGTH);
    int *current_states, *next_states, n_current_states, n_next_states;
    current_states = (int*) malloc (sizeof(int)*MAX_CURRENT_STATES);
    next_states = (int*)  malloc (sizeof(int)*MAX_CURRENT_STATES);
  	int current_start;
    int line_length = dlinelen[id];
		int new_line_length;
    for(int i=0; i<line_length; i++){
      line[i] = dbuffer[dlinestart[id] + i];
    }
		for(int nfa_id = 0; nfa_id < n_nfa; nfa_id++){
			current_start = 0;
			new_line_length = 0;
			int* nfadata = nfaset + nfaset[nfa_id*5 + 0];
			int is_global = nfaset[nfa_id*5 + 2];
			char* replacement_string = replacement_strings + nfaset[nfa_id*5 + 3];
			int replacement_length = nfaset[nfa_id*5 + 4];
			int match_len, last_final_state;
			do{
				n_current_states = 0;
				n_next_states = 0;
				match_len = 0;
				last_final_state = -1;
				add_to_next_states(nfadata, nfadata[0], current_states, &n_current_states);
        for(int i=0; i<n_current_states; i++){
          if(current_states[i] == nfadata[1]){
              last_final_state = match_len;
              break;
          }
        }
				while(n_current_states != 0 && (current_start + match_len < line_length)){
					for(int state_id = 0; state_id < n_current_states; state_id++){
						make_nfa_transition(nfadata, line[current_start + match_len], current_states[state_id], next_states, &n_next_states);
					}
          for(int i=0; i<n_next_states; i++){
            if(next_states[i] == nfadata[1]){
              last_final_state = match_len;
              break;
            }
          }
					int* temp = next_states;
					next_states = current_states;
					current_states = temp;
					n_current_states = n_next_states;
					n_next_states = 0;
					match_len++;
				}
				if(last_final_state != -1){
					current_start += last_final_state + 1;
					for(int i = 0; i < replacement_length; i++){
						new_line[new_line_length++] = replacement_string[i];
					}
					if(is_global == 0) {
						while(current_start < line_length){
							new_line[new_line_length++] = line[current_start++];
						}
						break;
					}
				} else if(current_start < line_length) {
					new_line[new_line_length++] = line[current_start];
					current_start++;
				}
			} while (current_start < line_length);
			char* temp = line;
			line = new_line;
			new_line = temp;
			line_length = new_line_length;
			new_line_length = 0;
		}
    __syncthreads();
    int i;
		for(i=0; i<line_length; i++){
      dbuffer[MAX_LINE_LENGTH*id + i] = line[i];
		}
    dbuffer[MAX_LINE_LENGTH*id + i] = '\0';
    free(line);
    free(new_line);
    free(current_states);
    free(next_states);
}
