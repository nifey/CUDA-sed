#include <unistd.h>
#include <fcntl.h>
#include <getopt.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <sys/mman.h>

#include "nfa.h"
#include "kernels.h"

#define VERSION "0.1"
#define DPRINTF(ARGS) if(debug==1) printf ARGS;

int debug = 0;

void print_help(char* command_name){
	printf("Usage: %s -f filename -e expression\n", command_name);
}

void print_version(){
	printf("CUDA-sed V" VERSION "\n");
}

NFAset* process_expression(char* expr, char** replacement_strings){
	char* expressions[MAX_EXPR];
	int count = 0;
	expressions[count++] = strtok(expr, ";");
	while(expressions[count-1] != NULL && count<MAX_EXPR){
		expressions[count++] = strtok(NULL, ";");
	}
	count--;
	NFA** nfas = (NFA**)malloc (sizeof(NFA*)*count);
	int* metadata = (int*) malloc (sizeof(int)*3*count);
	char* replacement[count];
	int replacement_total_length = 0;
	for(int expr_id=0;expr_id<count;expr_id++){
		DPRINTF(("Expression %d: %s\n",expr_id, expressions[expr_id]));
		char* command = strtok(expressions[expr_id], "/");
		if(command==NULL || command[0] != 's'){
			DPRINTF(("Only s command is supported\n"));
			exit(0);
		}
		char* search_regex = strtok(NULL, "/");
		replacement[expr_id] = strtok(NULL, "/");
		char* extra = strtok(NULL, "/");
		if(search_regex == NULL || replacement[expr_id] == NULL){
			DPRINTF(("Error in command syntax\n"));
			exit(0);
		}
		if(extra != NULL && index(extra, 'g')!=NULL){
			metadata[expr_id*3 + 0] = 1;
		} else {
			metadata[expr_id*3 + 0] = 0;
		}
		metadata[expr_id*3 + 1] = replacement_total_length;
		replacement_total_length += strlen(replacement[expr_id]);
		metadata[expr_id*3 + 2]= strlen(replacement[expr_id]);
		nfas[expr_id] = post2nfa(re2post(search_regex));
		if(debug == 1){
			char dotfile[10];
			sprintf(dotfile, "nfa%d.dot", expr_id);
			printDot(nfas[expr_id], dotfile);
		}
	}
	*replacement_strings = (char*) malloc (sizeof(char)*replacement_total_length);
	int index = 0;
	for(int expr_id = 0; expr_id<count; expr_id++){
		memcpy(*replacement_strings + index, replacement[expr_id], strlen(replacement[expr_id]));
		index += strlen(replacement[expr_id]);
	}
	*(*replacement_strings + index) = '\0';
	return NFA2NFAset(nfas, count, metadata);
}

int main(int argc, char* argv[]){
	char* shortopts = "hvcdf:e:b:";
	static const struct option longopts[] = {
		{"file", 1, NULL, 'f'},
		{"expression", 1, NULL, 'e'},
		{"debug", 0, NULL, 'd'},
		{"help", 0, NULL, 'h'},
		{"version", 0, NULL, 'v'},
		{"cpu", 0, NULL, 'c'},
		{"blocks", 0, NULL, 'b'},
		{NULL, 0, NULL, 0},
	};
	int opt;
	int cpu = 0, blocks = NUM_BLOCKS; 
	char *expression = NULL, *replacement_strings, *filename = NULL;
	while((opt = getopt_long(argc, argv, shortopts, longopts, NULL))!=EOF){
		switch (opt) {
			case 'f':
				DPRINTF(("Input File: %s\n", optarg));
				filename = strdup(optarg);
				break;
			case 'e':
				DPRINTF(("Expression: %s\n", optarg));
				expression = strdup(optarg);
				break;
			case 'd':
				debug = 1;
				DPRINTF(("Debug set to True\n"));
				break;
			case 'h':
				print_help(argv[0]);
				exit(0);
				break;
			case 'v':
				print_version();
				exit(0);
				break;
			case 'c':
				cpu = 1;
				break;
			case 'b':
				blocks = atoi(optarg);
				break;
			default:
				break;
		}
	}

	if(expression == NULL || filename == NULL){
		printf("Expression or filename required\n");
		exit(0);
	}
	// Process the expression and convert to NFA
	NFAset* nfaset = process_expression(expression, &replacement_strings);
	if(debug==1){
		printNFAset(nfaset, replacement_strings);
	}

	struct stat sb;
	int fd = open(filename, O_RDONLY);
	if(fstat(fd, &sb) == -1){ 
		printf("Error reading filestats\n");
		exit(0);
	}

	char* input_buffer = (char*) mmap(NULL, sb.st_size, PROT_READ, MAP_PRIVATE, fd, 0);
	if(input_buffer == MAP_FAILED){
		printf("Cannot MMap the input file\n");
		exit(0);
	}

	char* buffer_pointer = input_buffer;
	char* start_pointer = input_buffer;
	if(cpu==0){
		// Copy NFAset and replacement string to GPU
		int *dnfaset = copyNfasetToGPU(nfaset);
		char *dreplacement_strings = copyReplacementStringsToGPU(replacement_strings);

		int lastnewline = 1, num_lines;
		while(buffer_pointer < input_buffer + sb.st_size){
			// Open the File and split based on newline character
			int *linestart = (int*) malloc (sizeof(int)*blocks*THREADS_PER_BLOCK);
			int *linelen = (int*) malloc (sizeof(int)*blocks*THREADS_PER_BLOCK);
			start_pointer = buffer_pointer;
			int blockdatalen[blocks], oldblockno=0, sumblocklen = 0;
			for(int i=0; i<blocks; i++){
				blockdatalen[i] = 0;
			}
			num_lines = -1;
			for(int line_no=0; line_no<blocks*THREADS_PER_BLOCK; line_no++){
				if(buffer_pointer >= input_buffer + sb.st_size){
					linestart[line_no] = -1;
					linelen[line_no] = 0;
				} else {
					int line_size = 0;
					int block_no = (int)(line_no/THREADS_PER_BLOCK);
					if(oldblockno != block_no){
						sumblocklen+=blockdatalen[oldblockno];
						oldblockno = block_no;
					}
					if(block_no != 0){
						linestart[line_no] = block_no*THREADS_PER_BLOCK*MAX_LINE_LENGTH  + (buffer_pointer - start_pointer) - sumblocklen;
					} else {
						linestart[line_no] = block_no*THREADS_PER_BLOCK*MAX_LINE_LENGTH  + (buffer_pointer - start_pointer);
					}
					while(buffer_pointer < input_buffer + sb.st_size && *buffer_pointer != '\n'){
						buffer_pointer++;
						line_size++;
					}
					if(buffer_pointer < input_buffer + sb.st_size){
						blockdatalen[block_no] += line_size + 1;
					} else {
						blockdatalen[block_no] += line_size;
						lastnewline = 0;
					}
					linelen[line_no] = line_size;
					buffer_pointer++;
					if(buffer_pointer >= input_buffer + sb.st_size){
						num_lines = line_no + 1;
					}
				}
			}
			if(num_lines == -1){
				num_lines = blocks*THREADS_PER_BLOCK;
			}

			// Copy lines to GPU
			int *dlinestart, *dlinelen;
			char *dbuffer;
			copyLinesToGPU(linestart, linelen, start_pointer, blockdatalen, &dlinestart, &dlinelen, &dbuffer, blocks);
			start_pointer = buffer_pointer;
			free(linestart);
			free(linelen);

			// Process the lines
			int size = (nfaset->nfadata[(nfaset->n_nfa-1)*5+0] + nfaset->nfadata[(nfaset->n_nfa-1)*5+1]) * sizeof(int) + ((strlen(replacement_strings) * sizeof(char))/sizeof(int) + 1)*sizeof(int);
			processLines<<<blocks,THREADS_PER_BLOCK, size>>>(dnfaset, nfaset->nfadata[(nfaset->n_nfa-1)*5+0] + nfaset->nfadata[(nfaset->n_nfa-1)*5+1], nfaset->n_nfa, dreplacement_strings,strlen(replacement_strings), dlinestart, dlinelen, dbuffer);
			hipDeviceSynchronize();

			// Copy processed lines back to host and print the lines
			copyLinesBackAndPrint(dbuffer, num_lines, lastnewline, blocks);

		}

	} else {
		char* old_buffer_pointer;
		while(buffer_pointer < input_buffer + sb.st_size){
			char *line, *new_line;
			line = (char*) malloc (sizeof(char)*MAX_LINE_LENGTH);
			new_line = (char*) malloc (sizeof(char)*MAX_LINE_LENGTH);

			int line_size = 0;
			old_buffer_pointer = buffer_pointer;
			while(buffer_pointer < input_buffer + sb.st_size && *buffer_pointer != '\n'){
				buffer_pointer++;
				line_size++;
			}
			buffer_pointer++;
			memcpy(line, old_buffer_pointer, sizeof(char)*line_size);

			int current_start;
			int line_length = line_size;
			int new_line_length;
			for(int nfa_id = 0; nfa_id < nfaset->n_nfa; nfa_id++){
				current_start = 0;
				new_line_length = 0;
				int* current_states = (int*) malloc (sizeof(int)*MAX_CURRENT_STATES);
				int* next_states = (int*) malloc (sizeof(int)*MAX_CURRENT_STATES);
				int n_current_states, n_next_states;
				int* nfadata = nfaset->nfadata + nfaset->nfadata[nfa_id*5 + 0];
				int is_global = nfaset->nfadata[nfa_id*5 + 2];
				char* replacement_string = replacement_strings + nfaset->nfadata[nfa_id*5 + 3];
				int replacement_length = nfaset->nfadata[nfa_id*5 + 4];
				int match_len, last_final_state;
				do{
					n_current_states = 0;
					n_next_states = 0;
					match_len = 0;
					last_final_state = -1;
					add_to_next_states(nfadata, nfadata[0], current_states, &n_current_states);
					if(check_for_final_state(nfadata[1], current_states, n_current_states) == 1){
						last_final_state = match_len;
					}
					while(n_current_states != 0 && (current_start + match_len < line_length)){
						for(int state_id = 0; state_id < n_current_states; state_id++){
							make_nfa_transition(nfadata, line[current_start + match_len], current_states[state_id], next_states, &n_next_states);
						}
						if(check_for_final_state(nfadata[1], next_states, n_next_states) == 1){
							last_final_state = match_len;
						}
						int* temp = next_states;
						next_states = current_states;
						current_states = temp;
						n_current_states = n_next_states;
						n_next_states = 0;
						match_len++;
					}
					if(last_final_state != -1){
						current_start += last_final_state + 1;
						for(int i = 0; i < replacement_length; i++){
							new_line[new_line_length++] = replacement_string[i];
						}
						if(is_global == 0) {
							while(current_start < line_length){
								new_line[new_line_length++] = line[current_start++];
							}
							break;
						}
					} else if(current_start < line_length) {
						new_line[new_line_length++] = line[current_start];
						current_start++;
					}
				} while (current_start < line_length);
				char* temp = line;
				line = new_line;
				new_line = temp;
				line_length = new_line_length;
				new_line_length = 0;
			}
			for(int i=0; i<line_length; i++){
				printf("%c", line[i]);
			}
			printf("\n");

		}
	}

	munmap(input_buffer, sb.st_size);
}
